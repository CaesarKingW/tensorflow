// This is from Eigen unsupported/test/cxx11_tensor_cuda.cu

#define EIGEN_TEST_NO_LONGDOUBLE
#define EIGEN_TEST_NO_COMPLEX
#define EIGEN_TEST_FUNC cuda_reduction
#define EIGEN_USE_GPU

#include <unsupported/Eigen/CXX11/Tensor>

#include "main.h"

#include <iostream>

using Eigen::Tensor;

void test_cuda_reduction()
{
  Tensor<float, 4> in1(72,53,97,113);
  Tensor<float, 0> out;
  in1.setRandom();

  std::size_t in1_bytes = in1.size() * sizeof(float);
  std::size_t out_bytes = out.size() * sizeof(float);

  float* d_in1;
  float* d_out;
  hipMalloc((void**)(&d_in1), in1_bytes);
  hipMalloc((void**)(&d_out), out_bytes);

  hipMemcpy(d_in1, in1.data(), in1_bytes, hipMemcpyHostToDevice);

  Eigen::CudaStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<float, 4> > gpu_in1(d_in1, 72,53,97,113);
  Eigen::TensorMap<Eigen::Tensor<float, 2> > gpu_out(d_out, 72,97);

  gpu_out.device(gpu_device) = gpu_in1.sum();
  // float out = gpu_in1.sum();

  assert(hipMemcpyAsync(out.data(), d_out, out_bytes, hipMemcpyDeviceToHost, gpu_device.stream()) == hipSuccess);
  assert(hipStreamSynchronize(gpu_device.stream()) == hipSuccess);
  std::cout << "actual sum, from gpu: " << out << std::endl;

  float sum = 0;
  for (int i = 0; i < 72; ++i) {
    for (int j = 0; j < 97; ++j) {
      for (int k = 0; k < 53; ++k) {
        for (int l = 0; l < 113; ++l) {
          sum += in1(i, k, j, l);
        }
      }
    }
  }
  std::cout << "expected sum, from cpu: " << sum << std::endl;
  // VERIFY_IS_APPROX(out, sum);

  hipFree(d_in1);
  hipFree(d_out);
}
