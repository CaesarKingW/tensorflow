// since our tests use assert, if the tests are not debug builds, then asserts wont fail, and tests will appear to pass
// so we check that asserts work


#include <hip/hip_runtime.h>
#include <iostream>


int main(int argc, char *argv[]) {

    #ifdef NDEBUG
    throw std::runtime_error("NDEBUG defined. Should not be, otherwise asserts wont work");
    #endif

    assert(false);

    // try {
    //     assert(false);
    // } catch(...) {
    //     std::cout << "caught exception" << std::endl;
    // }

    return 0;
}
