
#include <hip/hip_runtime.h>
struct MyStruct {
    float floatvalue;
    int intvalue;
};

__device__ __host__ float sumStruct(struct MyStruct **p_structs, int N) {
    float sum = 0;
    for(int i = 0; i < N; i++) {
        struct MyStruct *mystruct = p_structs[i];
        sum += mystruct->floatvalue + float(mystruct->intvalue) * 3.5f;
    }
    return sum;
}

__global__ void mykernel(float *data, MyStruct *structs, int N) {
    data[0] = sumStruct(&structs, N);
    data[3] = sumStruct(&structs, 123);
    data[4] = sumStruct(&structs, 12300);
}
