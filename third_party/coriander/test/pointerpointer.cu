
#include <hip/hip_runtime.h>
// The code which is causing the pointer pointer address space error:

// %"struct.Eigen::half_impl::__half" = type { i16 }
// %"struct.Eigen::half_impl::half_base" = type { %"struct.Eigen::half_impl::__half" }
// %"struct.Eigen::half" = type { %"struct.Eigen::half_impl::half_base" }
// %"struct.Eigen::DSizes" = type { %"class.Eigen::array" }
// %"class.Eigen::array" = type { [1 x i32] }
// %"struct.Eigen::GpuDevice" = type <{ %"class.Eigen::StreamInterface"*, i32, [4 x i8] }>
// %"class.Eigen::StreamInterface" = type { i32 (...)** }
// %"struct.Eigen::TensorEvaluator.0" = type { %"struct.Eigen::half"*, %"struct.Eigen::DSizes", %"struct.Eigen::GpuDevice"* }
// %"struct.Eigen::TensorEvaluator.2" = type { %"struct.Eigen::half"*, %"struct.Eigen::DSizes", %"struct.Eigen::GpuDevice"* }
// %"struct.Eigen::TensorEvaluator.7" = type { %"struct.Eigen::internal::scalar_left", %"struct.Eigen::TensorEvaluator.2" }
// %"struct.Eigen::TensorEvaluator.6" = type { %"struct.Eigen::TensorEvaluator.0", %"struct.Eigen::TensorEvaluator.7" }

// ; Function Attrs: norecurse nounwind
// define weak_odr void @_ZN5Eigen8internal15EigenMetaKernelINS_15TensorEvaluatorIKNS_14TensorAssignOpINS_9TensorMapINS_6TensorINS_4halfELi1ELi1EiEELi16EEEKNS_18TensorCwiseUnaryOpINS0_11scalar_leftIS6_S6_NS0_13scalar_sum_opIS6_S6_EEEEKNS4_INS5_IKS6_Li1ELi1EiEELi16EEEEEEENS_9GpuDeviceEEEiEEvT_T0_(%"struct.Eigen::TensorEvaluator.6"* byval nocapture readonly align 8 %memcopied_eval, i32 %size) #1 comdat {
//   %1 = tail call i32 @llvm.ptx.read.ctaid.x() #6
//   %2 = tail call i32 @llvm.ptx.read.ntid.x() #6
//   %3 = mul i32 %2, %1
//   %4 = tail call i32 @llvm.ptx.read.tid.x() #6
//   %5 = add i32 %3, %4
//   %6 = tail call i32 @llvm.ptx.read.nctaid.x() #6
//   %7 = mul i32 %6, %2
//   %8 = getelementptr inbounds %"struct.Eigen::TensorEvaluator.6", %"struct.Eigen::TensorEvaluator.6"* %memcopied_eval, i64 0, i32 0, i32 0
//   %9 = load %"struct.Eigen::half"*, %"struct.Eigen::half"** %8, align 8


struct MyStruct {
    float floatvalue;
    int intvalue;
};

__device__ __host__ float sumStruct(struct MyStruct **p_structs, int N) {
    float sum = 0;
    for(int i = 0; i < N; i++) {
        struct MyStruct *mystruct = p_structs[i];
        sum += mystruct->floatvalue + float(mystruct->intvalue) * 3.5f;
    }
    return sum;
}

__global__ void mykernel(float *data, MyStruct *structs, int N) {
    data[0] = sumStruct(&structs, N);
    data[3] = sumStruct(&structs, 123);
    data[4] = sumStruct(&structs, 12300);
}

class HalfImpl {
public:
    short myshort;
};

class HalfBase : public HalfImpl {
};

class Half : public HalfBase {

};

class StreamInterface {
};

class GpuDevice {
public:
    int myint;
    StreamInterface *pstreamInterface;
};

class TensorEvaluator0 {
public:
    Half *phalf;
    GpuDevice gpuDevice;
};

class TensorEvaluator2 {
public:
    Half *phalf;
    GpuDevice gpuDevice;
};

class TensorEvaluator7 {
public:
    Half *phalf;
    TensorEvaluator2 te2;
};
class TensorEvaluator6 {
public:
    class TensorEvaluator0 te0;
    class TensorEvaluator7 te7;
};

// __device__ __host__ float getGpuDeviceValue(GpuDevice *devices, int N) {
//     float sum = 0;
//     for(int i = 0; i < N; i++) {
//         sum += devices[i].myfloat * (5.5f + i);
//     }
//     return sum;
// }

__device__ __host__ float getHalfValue(Half *half_, int a) {
    return (float)(half_[a].myshort + 123);
}

__global__ void myte6kernel(TensorEvaluator6 *structs, float *data, GpuDevice *gpudevices, int a, int b, int c) {
    Half *phalf = structs[a].te0.phalf;
    data[0] = getHalfValue(phalf, a);
    //gpudevices[b] = structs[a].te0.gpuDevice;
    // float sum = 0;
    // for(int i = 0; i < 1000; i++) {
    //     sum += getGpuDeviceValue(&structs[i].te0.gpuDevice, c + i);
    // }
    // data[1] = sum;
}
