#include "hip/hip_runtime.h"
struct Bar {
    float somefloat;
};

struct Foo {
    int myint;
    float somefloat;
};

__host__ __device__ float getFooValue(struct Foo foo) {
    return foo.somefloat;
}

class SomeClass {
public:
    SomeClass() {}
    SomeClass(float value) {
        foo.somefloat = value;
    }
    struct Foo foo = { 123, 1.23f };
    float getValue() {
        return foo.somefloat;
    }
    float getValue2() {
        return foo.somefloat * 100.0f + 2.0f;
    }
    float getValue3(float v) {
        return foo.somefloat * 100.0f * v + 2.0f;
    }
};

__host__ __device__ float getSomeClassValue(SomeClass someclass, SomeClass c2) {
    return someclass.foo.somefloat + c2.foo.somefloat;
}

__constant__ SomeClass myg1;
__constant__ SomeClass myg2(7.89f);
__constant__ struct Foo foo = {777, 11.0f};
__constant__ struct Bar bar = {12.0f};

__global__ void somekernel(float *data) {
    struct Foo bar = { 23, 0.0f };
    data[0] = bar.somefloat;
    data[1] = getFooValue(bar);
    SomeClass someclass;
    data[2] = someclass.foo.somefloat;
    SomeClass c2(3.45f);
    data[3] = getSomeClassValue(someclass, c2);
    data[4] = c2.getValue();
    data[5] = c2.getValue2();
    data[6] = c2.getValue3(data[7]);
    for(int i = 0; i < 10000; i++) {
        data[100+i] = c2.getValue3(data[10100 + i]);
    }
    data[8] = myg1.getValue3(data[9]);
    data[10] = myg2.getValue3(data[11]);
}

__global__ void somekernel2(float *data) {
    data[0] = getSomeClassValue(myg1, myg2);
}

__global__ void getFooValue(float *data, int *intdata) {
    data[0] = foo.somefloat;
    intdata[0] = foo.myint;
}

__global__ void getBarValue(float *data) {
    data[0] = bar.somefloat;
}
