
#include <hip/hip_runtime.h>
namespace mynamespace {
    namespace subnamespace {
        class Foo {
        public:
            int a;
            Foo() {

            }
            ~Foo() {

            }
            void somefunc() {

            }
        };
    }

    class Bar : public subnamespace::Foo {
    public:
        Foo foo;
        int b;
        Bar() {

        }
        ~Bar() {

        }
        void somefunc2() {

        }
    };

    template<typename T>
    class Templated : public subnamespace::Foo {
    public:
        T someattribute;
        void doSomething(T val) {

        }
    };
}

template<typename T>
__global__
void doSomething(mynamespace::Bar bar, mynamespace::Templated<T> t, T *data, int *int_data) {
    bar.a = 123;
    data[0] = t.someattribute;
    int_data[0] = bar.b;
}

__host__ void doSomething_host(mynamespace::Bar bar, mynamespace::Templated<float> t, float *data, int *int_data) {
    doSomething<<<dim3(32), dim3(32)>>>(bar, t, data, int_data);
}

__host__ void doSomething_host2(mynamespace::Bar bar, mynamespace::Templated<int> t, int *data, int *int_data) {
    doSomething<<<dim3(32), dim3(32)>>>(bar, t, data, int_data);
}

__host__ void doSomething_host2(mynamespace::Bar bar, mynamespace::Templated<char> t, char *data, int *int_data) {
    doSomething<<<dim3(32), dim3(32)>>>(bar, t, data, int_data);
}
