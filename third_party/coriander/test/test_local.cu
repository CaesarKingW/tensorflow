
#include <hip/hip_runtime.h>
__global__ void testLocal(float *data) {
    __shared__ float myshared[32];
    int tid = threadIdx.x;
    myshared[tid] = data[tid];
    data[0] = myshared[tid + 1];
}

__global__ void testLocal2(float *data) {
    __shared__ float myshared[64];
    int tid = threadIdx.x;
    myshared[tid] = data[tid];
    data[0] = myshared[tid + 1];
    myshared[tid + 1] = data[tid];
    data[1] = myshared[tid];
}
