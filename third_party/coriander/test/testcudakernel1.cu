#include "hip/hip_runtime.h"
#include <iostream>
using namespace std;

__device__ float bar(float a, float b) {
    return a + b;
}

__device__ void incrval(float *a) {
    *a += 3;
}

__global__ void foo(float *data) {
    data[0] = 123.0f;
}

__global__ void use_tid(float *data) {
    int tid = threadIdx.x;
    data[tid] = 123.0f;
}

__global__ void use_tid2(int *data) {
    int tid = threadIdx.x;
    data[tid] = data[tid] + tid;
}

__global__ void copy_float(float *a) {
    a[0] = a[1];
}

__global__ void use_blockid(float *data) {
    int blkid = blockIdx.x;
    data[blkid] = 123.0f;
}

__global__ void use_griddim(float *data) {
    int blkid = gridDim.x;
    data[blkid] = 123.0f;
}

__global__ void use_blockdim(float *data) {
    int blkid = blockDim.x;
    data[blkid] = 123.0f;
}

__host__ float someHostFunction(float input) {
    cout << "You called: someHostFunction()" << endl;
    return input * 100.0f;
}

__global__ void someops_float(float *data) {
    data[0] = data[1] - data[2];
    data[0] += data[1] / data[2];
    data[0] += data[1] * data[2];
    data[0] += log(data[1]);
    data[0] += exp(data[1]);
    data[0] += tanh(data[1]);
    data[0] -= sqrt(data[1]);
}

__global__ void someops_int(int *data) {
    data[0] = data[1] - data[2];
    data[0] += data[1] / data[2];
    data[0] += data[1] + data[2];
    data[0] += data[1] * data[2];
    data[0] += data[1] << data[2];
    data[0] += data[1] >> data[2];
}

__global__ void testbooleanops(int *data) {
    bool a = data[0] > 0;
    bool b = data[1] < 0;
    data[2] = (int)(a && b);
    data[3] = (int)(a || b);
    data[4] = (int)(!a);
}

__global__ void testcomparisons_int_signed(int *data) {
    data[5] = (int)(data[0] >= data[1]);
    data[6] = (int)(data[0] <= data[1]);
    data[7] = (int)(data[0] > data[1]);
    data[8] = (int)(data[0] < data[1]);
    data[9] = (int)(data[0] == data[1]);
    data[10] = (int)(data[0] != data[1]);
}

__global__ void testcomparisons_float(float *data) {
    data[5] = (data[0] >= data[1]);
    data[6] = (data[0] <= data[1]);
    data[7] = (data[0] > data[1]);
    data[8] = (data[0] < data[1]);
    data[9] = (data[0] == data[1]);
    data[10] = (data[0] != data[1]);
}

// // __global__ void testcomparisons_int_unsigned(unsigned int *data) {
// //     data[5] = (unsigned int)(data[0] >= data[1]);
// //     data[6] = (unsigned int)(data[0] <= data[1]);
// //     data[7] = (unsigned int)(data[0] > data[1]);
// //     data[8] = (unsigned int)(data[0] < data[1]);
// //     data[9] = (unsigned int)(data[0] == data[1]);
// //     data[10] = (unsigned int)(data[0] != data[1]);
// // }

__global__ void testsyncthreads(float *data) {
    int tid = threadIdx.x;
    data[tid] *= 2;
    syncthreads();
    data[tid + 1] += 2;
}

void myprintint(int value) {
    cout << "myprintint " << value << endl;
}

void myprintfloat(float value) {
    cout << "myprintfloat " << value << endl;
}

void myprintvoidstar(void *value) {
    cout << "myprintvoid* " << value << endl;
}

void mynop() {

}

__global__ void setValue(float *data, int idx, float value) {
    if(threadIdx.x == 0) {
        data[idx] = value;
    }
}

__host__ void launchSetValue(float *data, int idx, float value) {
    setValue<<<dim3(32, 1, 1), dim3(32, 1, 1)>>>(data, idx, value);
}

struct MyStruct {
    int x;
    float y;
};

__global__ void testStructs(MyStruct *structs, float *float_data, int *int_data) {
    int_data[0] = structs[0].x;
    float_data[0] = structs[0].y;
    float_data[1] = structs[1].y;
}

__global__ void testFloat4(float4 *data) {
    float4 myregister4 = data[1];
    float *myregisterfloat = (float *)&myregister4;
    myregisterfloat[1] = myregisterfloat[2] * myregisterfloat[3];
    data[0] = myregister4;
}

__global__ void testFloat4_test2(float4 *data) {
    data[0] = data[1];
}

// __global__ void testFloat4_test3(float4 *data) {
//     float4 privateFloats[32];
//     for(int i = 0; i < 32; i++) {
//         privateFloats[i] = data[i];
//     }
//     for(int i = 0; i < 32; i++) {
//         data[i + 1] = privateFloats[i];
//     }
// }

__global__ void testLocal(float *data) {
    __shared__ float myshared[32];
    int tid = threadIdx.x;
    myshared[tid] = data[tid];
    data[0] = myshared[tid + 1];
}

__global__ void testLocal2(float *data) {
    __shared__ float myshared[64];
    int tid = threadIdx.x;
    myshared[tid] = data[tid];
    data[0] = myshared[tid + 1];
    myshared[tid + 1] = data[tid];
    data[1] = myshared[tid];
}

__global__ void testArray(float *data) {
    float privateFloats[32];
    for(int i = 0; i < 32; i++) {
        privateFloats[i] = data[i * 3];
    }
    for(int i = 0; i < 32; i+= 2) {
        data[i + 1] = privateFloats[i];
    }
}

__global__ void testmemcpy(float *data) {
    float privateFloats[32];
    for(int i = 0; i < 32; i++) {
        privateFloats[i] = data[i];
    }
    for(int i = 0; i < 32; i+= 2) {
        data[i] = privateFloats[i];
    }
}

__device__ float4 getfloat4(float a) {
    float4 res;
    res.x = a;
    res.y = a + 1;
    res.z = a + 2.5f;
    return res;
}

// attempting to generate an extractvalue instruciton, but failing :-P
__device__ float getfloat4ElementSum(float a, int e0, int e1) {
    float4 res = getfloat4(a);
    float sum = 0;
    sum += ((float *)&res)[e0];
    sum += ((float *)&res)[e1];
    // float4 res;
    // res.x = a;
    // res.y = a + 1;
    // res.z = a + 2.5f;
    // return res;
    return sum;
}

// __global__ void testFloat4_insertvalue(float4 *data, float *data2, int N) {
//     float4 res = getfloat4(data2[0]);
//     data[0] = res;
// }

struct hasArray {
    int foo[4];
};

// __global__ void useHasArray(hasArray *data) {
//     data[0].foo[0] = data[1].foo[2];
// }

__device__ float declaredAfterUse(float val1, float val2);

__global__ void usesForwardDeclaration(float *data) {
    data[0] = declaredAfterUse(data[1], data[2]);
}

__device__ float declaredAfterUse(float val1, float val2) {
    return val1 * val2;
}
